#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda-backend.h"

void create_dev_buffers_cuda(sgDataBuf* source, sgDataBuf* target, 
                            sgIndexBuf* si, sgIndexBuf *ti, 
                            size_t block_len){
    hipMalloc((void **)&(source->dev_ptr), source->size);
    hipMalloc((void **)&(target->dev_ptr), target->size);
}
