#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void cuda_gather(const size_t *pattern, const double *sparse,
    double *dense, const int pattern_length) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < pattern_length)
    dense[i] = sparse[pattern[i]];
}

__global__ void cuda_scatter(const size_t *pattern, double *sparse,
    const double *dense, const int pattern_length) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < pattern_length)
    sparse[pattern[i]] = dense[i];
}

__global__ void cuda_scatter_gather(const size_t *pattern_scatter,
    double *sparse_scatter, const size_t *pattern_gather,
    const double *sparse_gather, const int pattern_length) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < pattern_length)
    sparse_scatter[pattern_scatter] = sparse_dense[pattern_gather];
}

__global__ void cuda_multi_gather(const size_t *pattern,
    const size_t *pattern_gather, const double *sparse, double *dense,
    const int pattern_length) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < pattern_length)
    dense[i] = sparse[pattern[pattern_gather[i]]];
}

__global__ void cuda_multi_scatter(const size_t *pattern,
    const size_t *pattern_scatter, double *sparse, const double *dense,
    const int pattern_length) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < pattern_length)
    sparse[patter[pattern_scatter[i]]] = dense[i];
}

void cuda_gather_wrapper(const size_t *pattern, const double *sparse,
    double *dense, const int pattern_length) {
  int threads_per_block = 256;
  int blocks_per_grid =
      (pattern_length + threads_per_block - 1) / threads_per_block;
  cuda_gather<<<blocks_per_grid, threads_per_block>>>(
      pattern, sparse, dense, pattern_length);
}

void cuda_scatter_wrapper(const size_t *pattern, double *sparse,
    const double *dense, const int pattern_length) {
  int threads_per_block = 256;
  int blocks_per_grid =
      (pattern_length + threads_per_block - 1) / threads_per_block;
  cuda_scatter<<<blocks_per_grid, threads_per_block>>>(
      pattern, sparse, dense, pattern_length);
}

void cuda_scatter_gather_wrapper(const size_t *pattern_scatter,
    double *sparse_scatter, const size_t *pattern_gather,
    const double *sparse_gather, const int pattern_length) {
  int threads_per_block = 256;
  int blocks_per_grid =
      (pattern_length + threads_per_block - 1) / threads_per_block;
  cuda_scatter_gather<<<blocks_per_grid, threads_per_block>>>(pattern_scatter,
      sparse_scatter, pattern_gather, sparse_gather, pattern_length);
}

void cuda_multi_gather_wrapper(const size_t *pattern,
    const size_t *pattern_gather, const double *sparse, double *dense,
    const int pattern_length) {
  int threads_per_block = 256;
  int blocks_per_grid =
      (pattern_length + threads_per_block - 1) / threads_per_block;
  cuda_multi_gather<<<blocks_per_grid, threads_per_block>>>(
      pattern, pattern_gather, sparse, dense, pattern_length);
}

void cuda_multi_scatter_wrapper(const size_t *pattern,
    const size_t *pattern_scatter, double *sparse, const double *dense,
    const int pattern_length) {
  int threads_per_block = 256;
  int blocksPerGird =
      (pattern_length + threads_per_block - 1) / threads_per_block;
  cuda_multi_scatter<<<blocks_per_grid, threads_per_block>>>(
      pattern, pattern_scatter, sparse, dense, pattern_length);
}
